extern "C++" {
#include "rnn_layer.h"
#include "gemm.h"
}

namespace nebula {

extern "C++" void rnn_layer_t::_forward_() {
    network->batch_size /= network->time_step;

    if(network->run_type == TRAIN_RUN) {
        hipMemset(delta_dev, 0.0, output_size * network->batch_size * sizeof(float));
        hipMemcpy(prev_state_dev, state_dev, 
                   output_size * network->batch_size * sizeof(float), hipMemcpyDeviceToDevice);
    }
    
    for(unsigned step = 0; step < network->time_step; step++) {

        input_gate->_forward_();

        // Forward propagation of hidden layer in rnn layer.
        if(step) {hidden_gate->_forward_(state_dev);}
        else {hidden_gate->_forward_();}

        hipMemset(state_dev, 0.0, output_size * network->batch_size * sizeof(float));

        const float alpha = 1.0;

        //Add input gate and hidden gate.
#ifdef CUSTOM_BLAS
        _axpy_(output_size * network->batch_size, alpha, 
               input_gate->output_data_dev, 1, state_dev, 1);
        _axpy_(output_size * network->batch_size, alpha, 
               hidden_gate->output_data_dev, 1, state_dev, 1);
#else
        hipblasSaxpy(network->cublas_handle, output_size * network->batch_size, &alpha, 
                    input_gate->output_data_dev, 1, state_dev, 1);
        hipblasSaxpy(network->cublas_handle, output_size * network->batch_size, &alpha, 
                    hidden_gate->output_data_dev, 1, state_dev, 1); 
#endif
        output_gate->_forward_(state_dev);
        // Jump to next time step.
        if(prev_layer) {
            prev_layer->output_data_dev += prev_layer->output_size * network->batch_size;
        }
        else {
            network->input_data_dev += network->input_size * network->batch_size;
        }
    
        input_gate->_increment_(1);
        hidden_gate->_increment_(1);
        output_gate->_increment_(1);
    }
    
    if(prev_layer) { prev_layer->output_data_dev -= prev_layer->output_size * network->batch_size * network->time_step; }
    else { network->input_data_dev -= network->input_size * network->batch_size * network->time_step;}

    input_gate->_increment_(-network->time_step);
    hidden_gate->_increment_(-network->time_step);
    output_gate->_increment_(-network->time_step);

    network->batch_size *= network->time_step;

}


extern "C++" void rnn_layer_t::_backward_() {

    network->batch_size /= network->time_step;
    const float alpha = 1.0; 
    connected_layer_t *t_input_gate = input_gate;
    connected_layer_t *t_hidden_gate = hidden_gate;


    t_input_gate->_increment_(network->time_step);
    t_hidden_gate->_increment_(network->time_step);
    
    if(prev_layer) { prev_layer->output_data_dev += prev_layer->output_size * network->batch_size * network->time_step; }
    else { network->input_data_dev += network->input_size * network->batch_size * network->time_step; }

    for(int step = network->time_step -1; step >=0; step--) {

        t_input_gate->_increment_(-1);
        t_hidden_gate->_increment_(-1);

        if(prev_layer) { prev_layer->output_data_dev -= prev_layer->output_size * network->batch_size; }
        else { network->input_data_dev -= network->input_size * network->batch_size; }

        hipMemset(state_dev, 0.0, output_size * network->batch_size * sizeof(float));
#ifdef _CUSTOM_BLAS
        _axpy_(output_size * network->batch_size, alpha, t_input_gate->output_data_dev, 1, state_dev, 1);
        _axpy_(output_size * network->batch_size, alpha, t_hidden_gate->output_data_dev, 1, state_dev, 1);
#else
        hipblasSaxpy(network->cublas_handle, output_size * network->batch_size, &alpha, 
                    t_input_gate->output_data_dev, 1, state_dev, 1);
        hipblasSaxpy(network->cublas_handle, output_size * network->batch_size, &alpha, 
                    t_hidden_gate->output_data_dev, 1, state_dev, 1); 
#endif

        if(step == 0) {
            hipMemcpy(state_dev, prev_state_dev, 
                       output_size * network->batch_size * sizeof(float), hipMemcpyDeviceToDevice);
        }
        else {
            hipMemset(state_dev, 0.0, output_size * network->batch_size * sizeof(float));
#ifdef CUSTOM_BLAS
            _axpy_(output_size * network->batch_size, alpha, t_input_gate->output_data_dev - output_size * network->batch_size, 1, state_dev, 1);
            _axpy_(output_size * network->batch_size, alpha, t_hidden_gate->output_data_dev - output_size * network->batch_size, 1, state_dev, 1);
#else
            hipblasSaxpy(network->cublas_handle, output_size * network->batch_size, &alpha, 
                        t_input_gate->output_data_dev - output_size * network->batch_size, 1, state_dev, 1);
            hipblasSaxpy(network->cublas_handle, output_size * network->batch_size, &alpha, 
                        t_hidden_gate->output_data_dev - output_size * network->batch_size, 1, state_dev, 1);
#endif
        }

        hipMemcpy(t_input_gate->delta_dev, t_hidden_gate->delta_dev, 
                   output_size * network->batch_size * sizeof(float), hipMemcpyDeviceToDevice);

        t_hidden_gate->_backward_(state_dev, ((step > 0) ? t_hidden_gate->delta_dev - output_size * network->batch_size : 0));
         
        t_input_gate->_backward_();
    }
    hipMemset(state_dev, 0.0, output_size * network->batch_size * sizeof(float));
#ifdef CUSTOM_BLAS
    _axpy_(output_size * network->batch_size, alpha, t_input_gate->output_data_dev, 1, state_dev, 1);
    _axpy_(output_size * network->batch_size, alpha, t_hidden_gate->output_data_dev, 1, state_dev, 1);
#else
    hipblasSaxpy(network->cublas_handle, output_size * network->batch_size, 
                &alpha, t_input_gate->output_data_dev, 1, state_dev, 1);
    hipblasSaxpy(network->cublas_handle, output_size * network->batch_size, 
                &alpha, t_hidden_gate->output_data_dev, 1, state_dev, 1);
#endif

    network->batch_size *= network->time_step;
}

extern "C++" void rnn_layer_t::_update_() {
    input_gate->_update_();
    hidden_gate->_update_();
}

}
// End of namespace nebula
